#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <thrust/logical.h>

namespace cudf {
namespace strings {

/**
   * Parses this UTF8String(trimmed if needed) to INT8/16/32/64...
   *
   * Note that, in this method we accumulate the result in negative format, and convert it to
   * positive format at the end, if this string is not started with '-'. This is because min value
   * is bigger than max value in digits, e.g. Long.MAX_VALUE is '9223372036854775807' and
   * Long.MIN_VALUE is '-9223372036854775808'.
   *
   * This code is mostly copied from LazyLong.parseLong in Hive.
   *
 * @param d_str String to check.
 * @param allow_decimal Decimal format or not
 * @param min_value min_value that corresponds to the type that is checking
 * @return true if string has valid integer characters
   */
__device__ bool is_valid_fixed_point(string_view const& d_str, bool allow_decimal, long min_value)
{
  int offset = 0;
  size_type bytes = d_str.size_bytes();
  const char* data    = d_str.data();
  while (offset < bytes && data[offset] == ' ') ++offset;
  if (offset == bytes)  return false;

  int end = bytes - 1;
  while (end > offset && data[end] == ' ') --end;

  char c_sign = data[offset];
  const bool negative = c_sign == '-';
  if (negative || c_sign == '+'){
    if (end - offset == 0)  return false;
    ++offset;
  }

  const char separator = '.';
  const int radix = 10;
  const long stop_value = min_value / radix;
  long result = 0;

  while (offset <= end) {
    const char c = data[offset];
    ++offset;
    // We allow decimals and will return a truncated integral in that case.
    // Therefore we won't throw an exception here (checking the fractional
    // part happens below.
    if (c == separator && allow_decimal)  break;

    int digit;
    if (c >= '0' && c <= '9'){
      digit = c - '0';
    } else {
      return false;
    }

    // We are going to process the new digit and accumulate the result. However, before doing
    // this, if the result is already smaller than the stopValue(Long.MIN_VALUE / radix), then
    // result * 10 will definitely be smaller than minValue, and we can stop.
    if (result < stop_value)  return false;

    result = result * radix - digit;

    // Since the previous result is less than or equal to stopValue(Long.MIN_VALUE / radix), we
    // can just use `result > 0` to check overflow. If result overflows, we should stop.
    if (result > 0) return false;
  }
  // This is the case when we've encountered a decimal separator. The fractional
  // part will not change the number, but we will verify that the fractional part
  // is well formed.
  while (offset <= end) {
    char currentByte = data[offset];
    if (currentByte < '0' || currentByte > '9') return false;
    ++offset;
  }

  if (!negative) {
    result = -result;
    if (result < 0) return false;
  }

  return true;
}

namespace detail {

std::unique_ptr<column> is_valid_fixed_point(
  strings_column_view const& strings,
  bool allow_decimal,
  data_type input_type,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;
  auto d_allow_decimal = allow_decimal;

  // ready a min_value corresponds to the input type in order to check overflow
  long d_min_value = 0;
  switch (input_type.id()) {
    case type_id::INT8: d_min_value = -128;
    case type_id::INT16: d_min_value = -32768;
    case type_id::INT32: d_min_value = -2147483648;
    case type_id::INT64: d_min_value = -9223372036854775808;
    default: CUDF_FAIL("Unsupported current data type check when convert string type");
  }

  // create output column
  auto results   = make_numeric_column(data_type{type_id::BOOL8},
                                     strings.size(),
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto d_results = results->mutable_view().data<bool>();
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings.size()),
                    d_results,
                    [d_column,d_allow_decimal,d_min_value] __device__(size_type idx) {
                      if (d_column.is_null(idx)) return false;
                      return strings::is_valid_fixed_point(d_column.element<string_view>(idx), d_allow_decimal, d_min_value);
                    });
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace detail

// external API

std::unique_ptr<column> is_valid_fixed_point(strings_column_view const& strings,
                                          bool allow_decimal,
                                          data_type input_type,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_valid_fixed_point(strings, allow_decimal, input_type, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf

