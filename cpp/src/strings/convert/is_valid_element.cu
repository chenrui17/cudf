#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, Baidu CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/logical.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {
/**
 * Check whether the string is valid when convert string to signed integers,
 * like INT8/16/32/64. For example, if allow_decimal is true, then strings 
 * `['1.23', '123']` will return `[true, true]`.
 * If `allow_decimal` is false, then this function will return `[false, true]`.
 * 
 * Note that, in this method we accumulate the result in negative format, and convert it to
 * positive format at the end, if this string is not started with '-'. This is because min value
 * is bigger than max value in digits, e.g. Long.MAX_VALUE is '9223372036854775807' and
 * Long.MIN_VALUE is '-9223372036854775808'.
 *
 * This code is heavily based off of LazyLong.parseLong from Hive, but updated for C++.
 *
 * @param d_str String to check.
 * @param allow_decimal whether we allow the data is Decimal type or not.
 * @param min_value min_value that corresponds to the type that is checking.
 * @return true if string has valid integer characters or decimal characters.
 */
__device__ bool is_valid_element(string_view const& d_str, bool allow_decimal, long min_value)
{
  int offset = 0;
  size_type bytes = d_str.size_bytes();
  const char* data    = d_str.data();
  // strip leading white space
  while (offset < bytes && data[offset] == ' ') ++offset;
  if (offset == bytes)  return false;

  int end = bytes - 1;
  // strip trailing white space
  while (end > offset && data[end] == ' ') --end;

  char c_sign = data[offset];
  const bool negative = c_sign == '-';
  if (negative || c_sign == '+'){
    if (end - offset == 0)  return false;
    ++offset;
  }

  const char separator = '.';
  const int radix = 10;
  const long stop_value = min_value / radix;
  long result = 0;

  while (offset <= end) {
    const char c = data[offset];
    ++offset;
    // We allow decimals and will return a truncated integral in that case.
    // Therefore we won't throw an exception here (checking the fractional
    // part happens below).
    if (c == separator && allow_decimal)  break;

    int digit;
    if (c >= '0' && c <= '9'){
      digit = c - '0';
    } else {
      return false;
    }

    // We are going to process the new digit and accumulate the result. However, 
    // before doing this, if the result is already smaller than the stop_value which is
    // (std::numeric_limits<data_type>::min() / radix), then result * 10 will definitely 
    // be smaller than the min_value, and we can stop.
    if (result < stop_value)  return false;

    result = result * radix - digit;

    // Since the previous result is less than or equal to stopValue which is 
    // (std::numeric_limits<data_type>::min() / radix), we can just use `result > 0` 
    // to check overflow. If result overflows, we should stop.
    if (result > 0) return false;
  }
  // This is the case when we've encountered a decimal separator. The fractional
  // part will not change the number, but we will verify that the fractional part
  // is well formed.
  if (offset <= end && thrust::any_of(thrust::seq,
                                      data+offset,
                                      data+end,
                                      [] (char ch) {
                                        return (ch<'0' || ch>'9');
                                      }))
    return false;

  if (!negative) {
    result = -result;
    if (result < 0) return false;
  }

  return true;
}

} //namespace

/**
 * @brief The dispatch functions return the min value of the input data type
 * used for checking overflow.
 *
 * The output is the min value of specified type.
 */
struct min_value_of_type{
  template <typename T>
  long operator()()
  { 
    CUDF_FAIL("Unsupported current data type check."); 
  }
};

template <>
long min_value_of_type::operator()<int8_t>() { return std::numeric_limits<int8_t>::min(); }

template <>
long min_value_of_type::operator()<int16_t>() { return std::numeric_limits<int16_t>::min(); }

template <>
long min_value_of_type::operator()<int32_t>() { return std::numeric_limits<int32_t>::min(); }

template <>
long min_value_of_type::operator()<int64_t>() { return std::numeric_limits<int64_t>::min(); }

std::unique_ptr<column> is_valid_element(
  strings_column_view const& strings,
  bool allow_decimal,
  data_type input_type,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;
  auto d_allow_decimal = allow_decimal;

  // ready a min_value corresponds to the input type in order to check overflow
  long d_min_value = cudf::type_dispatcher(input_type, min_value_of_type{}) ;

  // create output column
  auto results   = make_numeric_column(data_type{type_id::BOOL8},
                                     strings.size(),
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto d_results = results->mutable_view().data<bool>();
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings.size()),
                    d_results,
                    [d_column,d_allow_decimal,d_min_value] __device__(size_type idx) {
                      if (d_column.is_null(idx)) return false;
                      return is_valid_element(d_column.element<string_view>(idx), d_allow_decimal, d_min_value);
                    });
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace detail

// external API

std::unique_ptr<column> is_valid_element(strings_column_view const& strings,
                                          bool allow_decimal,
                                          data_type input_type,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_valid_element(strings, allow_decimal, input_type, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf

